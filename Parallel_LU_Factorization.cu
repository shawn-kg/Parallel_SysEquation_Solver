#include "hip/hip_runtime.h"
/**
 * @file Parallel_LU_Factorization.cu
 * @author Shawn George
 * @author Adelin Owona
 * @author Michael Lenyszn
 * @author Miles Corn
 * @brief This file performs LU factorization on a matrix using partial pivoting
 * in parallel
 * @version 0.1
 * @date 2023-04-05
 *
 * @copyright Copyright (c) 2023
 *
 */

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include <random>

using namespace std;



#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include <random>
#include "clockcycle.h"

using namespace std;

__global__ void check_matrix_equivalence(double** A, double** B, bool* equal,
                                         int dimension) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  *equal = true;

  for (int r = index; r < dimension; r += stride) {
    for (int c = 0; c < dimension; c++) {
      if (fabs(A[r][c] - B[r][c]) > 0.0001) {
        printf("A[%d][%d] = %f, B[%d][%d] = %f\n", r, c, A[r][c], r, c,
               B[r][c]);
        *equal = false;
        return;
      }
    }
  }
}

// kernel to perform matrix multiplication
__global__ void matrix_mult(double** A, double** B, double** C, int dimension) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int r = index; r < dimension; r += stride) {
    for (int c = 0; c < dimension; c++) {
      C[r][c] = 0;
      for (int k = 0; k < dimension; k++) {
        C[r][c] += A[r][k] * B[k][c];
      }
    }
  }
}

// kernel to swap rows in U
__global__ void swap_rows_U(int row, int max_index, int col, int dimension,
                            double** U) {
  double rowholder;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int k = col + index; k < dimension; k += stride) {
    rowholder = U[row][k];
    U[row][k] = U[max_index][k];
    U[max_index][k] = rowholder;
  }
}

// kernel to swap rows in L
__global__ void swap_rows_L(int row, int max_index, int col, double** L) {
  double rowholder;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int k = index; k < col - 1; k += stride) {
    rowholder = L[row][k];
    L[row][k] = L[max_index][k];
    L[max_index][k] = rowholder;
  }
}

// kernel to swap rows of P
__global__ void swap_rows_P(int row, int max_index, int dimension, double** P) {
  double rowholder;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int k = index; k < dimension; k += stride) {
    rowholder = P[row][k];
    P[row][k] = P[max_index][k];
    P[max_index][k] = rowholder;
  }
}

// kernel to perform row operations
__global__ void row_ops_kernel(int col, int dimension, double** L, double** U) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int r = col + 1 + index; r < dimension; r += stride) {
    L[r][col] = U[r][col] / U[col][col];

    __syncthreads();  // make sure all threads have computed L[r][col]

    // distribute this loop across the y-dimension of the threadblock
    for (int k = col + threadIdx.y; k < dimension; k += blockDim.y) {
      U[r][k] = U[r][k] - L[r][col] * U[col][k];
    }
  }
}

// function to print a matrix
void print_matrix(double** matrix, int dimension) {
  for (int r = 0; r < dimension; r++) {
    for (int c = 0; c < dimension; c++) {
      printf("%f ", matrix[r][c]);
    }
    printf("\n");
  }
}

void LU_fact(double** matrix, double** L, double** U, double** P,
             int dimension) {
  // make sure that P,L = I and U = matrix
  for (int r = 0; r < dimension; r++) {
    for (int c = 0; c < dimension; c++) {
      if (r == c) {
        L[r][c] = 1;
        P[r][c] = 1;
      } else {
        L[r][c] = 0;
        P[r][c] = 0;
      }
      U[r][c] = matrix[r][c];
    }
  }

  // begin factorization with partial pivoting
  for (int c = 0; c < dimension - 1; c++) {
    double max = fabs(U[c][c]);
    int max_index = c;
    // find the max for the partial pivot
    for (int r = c; r < dimension - 1; r++) {
      if (fabs(U[r][c]) > max) {
        max = fabs(U[r][c]);
        max_index = r;
      }
    }

    swap_rows_U<<<1024, 1024>>>(c, max_index, c, dimension, U);

    swap_rows_L<<<1024, 1024>>>(c, max_index, c, L);

    swap_rows_P<<<1024, 1024>>>(c, max_index, dimension, P);

    row_ops_kernel<<<1024, 1024>>>(c, dimension, L, U);
    hipDeviceSynchronize();
  }
}



// cuda kernel to generate a strictly diagonally dominant matrix
__global__ void generateSDDMatrix(double** matrix, int n) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < n && col < n) {
    if (row == col) {
      int sum = 0;
      for (int i = 0; i < n; i++) {
        if (i != row) {
          sum += matrix[row][i];
        }
      }
      matrix[row][col] = sum + 1;
    }
  }
}

__global__ void rand_init(hiprandState* state) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(1337, idx, 0, &state[idx]);
}

__global__ void generateRandomValues(double** matrix, int n,
                                     hiprandState* state) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < n && col < n) {
    hiprandState localState = state[row * n + col];
    double rand = hiprand_uniform_double(&localState);
    state[row * n + col] = localState;

    // use rand to generate random values
    matrix[row][col] = rand * 9 + 1;
  }
}

int main(int argc, char* argv[]) {
  // initialize matrix A using hipMallocManaged
<<<<<<< HEAD
  int dimension = 50;
=======
  int dimension = 800;
>>>>>>> fc75f8d1905ba362255309fe43a1ab7f3e2539fc
  double** A;
  double** L;
  double** U;
  double** P;
  double** LU;
  double** PA;
  hiprandState* state;
  bool* equal;

  unsigned long long start_time = clock_now(); // used to time functions
  unsigned long long end_time = clock_now();
  double cycles_per_second = 512000000;

  hipMallocManaged(&A, dimension * sizeof(double*));
  hipMallocManaged(&L, dimension * sizeof(double*));
  hipMallocManaged(&U, dimension * sizeof(double*));
  hipMallocManaged(&P, dimension * sizeof(double*));
  hipMallocManaged(&LU, dimension * sizeof(double*));
  hipMallocManaged(&PA, dimension * sizeof(double*));
  hipMallocManaged(&equal, sizeof(bool));
  hipMallocManaged(&state, dimension * dimension * sizeof(hiprandState));

  for (int r = 0; r < dimension; r++) {
    hipMallocManaged(&A[r], dimension * sizeof(double));
    hipMallocManaged(&L[r], dimension * sizeof(double));
    hipMallocManaged(&U[r], dimension * sizeof(double));
    hipMallocManaged(&P[r], dimension * sizeof(double));
    hipMallocManaged(&LU[r], dimension * sizeof(double));
    hipMallocManaged(&PA[r], dimension * sizeof(double));
  }



  // initialize hiprand state
  rand_init<<<dimension, dimension>>>(state);

  int block_size = 32;

  dim3 grid_size((dimension + block_size - 1) / block_size,
                 (dimension + block_size - 1) / block_size);
  dim3 blocksize(block_size, block_size);

  // initialize A to be a random matrix
  generateRandomValues<<<grid_size, blocksize>>>(A, dimension, state);

  // generate a strictly diagonally dominant matrix
  generateSDDMatrix<<<grid_size, blocksize>>>(A, dimension);
  hipDeviceSynchronize();

  // print A
  // printf("A = \n");
  // print_matrix(A, dimension);

  // LU factorization
  start_time = clock_now();
  LU_fact(A, L, U, P, dimension);
  end_time = clock_now();
  double time_elapsed = (double) ((end_time-start_time)/cycles_per_second);

  // // print results
  // printf("L = \n");
  // print_matrix(L, dimension);

  // printf("\nU = \n");
  // print_matrix(U, dimension);

  // compute LU and PA
  matrix_mult<<<dimension, dimension>>>(L, U, LU, dimension);
  // hipDeviceSynchronize();
  matrix_mult<<<dimension, dimension>>>(P, A, PA, dimension);
  hipDeviceSynchronize();

  // check if LU = PA using check_matrix_equivalence
  check_matrix_equivalence<<<dimension, dimension>>>(LU, PA, equal, dimension);
  hipDeviceSynchronize();

  // print number of dimensions
  printf("Dimension: %d\n", dimension);

  // print results
  if (*equal) {
    printf("\nLU = PA\n");
  } else {
    printf("\nLU != PA\n");
  }

  printf("Time elapsed: %f seconds\n", time_elapsed);

  // printf("LU = \n");
  // print_matrix(LU, dimension);
  // printf("\nPA = \n");
  // print_matrix(PA, dimension);

  // free memory
  for (int r = 0; r < dimension; r++) {
    hipFree(A[r]);
    hipFree(L[r]);
    hipFree(U[r]);
    hipFree(P[r]);
    hipFree(LU[r]);
    hipFree(PA[r]);
  }

  hipFree(A);
  hipFree(L);
  hipFree(U);
  hipFree(P);
  hipFree(LU);
  hipFree(PA);
  hipFree(equal);
  hipFree(state);

  return 0;
}