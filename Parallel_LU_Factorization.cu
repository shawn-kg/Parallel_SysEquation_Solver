/**
 * @file Parallel_LU_Factorization.cu
 * @author Shawn George
 * @author Adelin Owona
 * @author Michael Lenyszn
 * @author Miles Corn
 * @brief This file performs LU factorization on a matrix using partial pivoting
 * in parallel
 * @version 0.1
 * @date 2023-04-05
 *
 * @copyright Copyright (c) 2023
 *
 */

#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include <random>

using namespace std;

__global__ void check_matrix_equivalence(double** A, double** B, bool* equal,
                                         int dimension) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  *equal = true;

  for (int r = index; r < dimension; r += stride) {
    for (int c = 0; c < dimension; c++) {
      if (fabs(A[r][c] - B[r][c]) > 0.0001) {
        printf("A[%d][%d] = %f, B[%d][%d] = %f",r,c,A[r][c],r,c,B[r][c]);
        *equal = false;
        return;
      }
    }
  }
}

// kernel to perform matrix multiplication
__global__ void matrix_mult(double** A, double** B, double** C, int dimension) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int r = index; r < dimension; r += stride) {
    for (int c = 0; c < dimension; c++) {
      C[r][c] = 0;
      for (int k = 0; k < dimension; k++) {
        C[r][c] += A[r][k] * B[k][c];
      }
    }
  }
}

// kernel to swap rows in U
__global__ void swap_rows_U(int row, int max_index, int col, int dimension,
                            double** U) {
  double rowholder;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int k = col + index; k < dimension; k += stride) {
    rowholder = U[row][k];
    U[row][k] = U[max_index][k];
    U[max_index][k] = rowholder;
  }
}

// kernel to swap rows in L
__global__ void swap_rows_L(int row, int max_index, int col, double** L) {
  double rowholder;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int k = index; k < col - 1; k += stride) {
    rowholder = L[row][k];
    L[row][k] = L[max_index][k];
    L[max_index][k] = rowholder;
  }
}

// kernel to swap rows of P
__global__ void swap_rows_P(int row, int max_index, int dimension, double** P) {
  double rowholder;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int k = index; k < dimension; k += stride) {
    rowholder = P[row][k];
    P[row][k] = P[max_index][k];
    P[max_index][k] = rowholder;
  }
}

// kernel to perform row operations
__global__ void row_ops_kernel(int col, int dimension, double** L, double** U) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int r = col + 1 + index; r < dimension; r += stride) {
    L[r][col] = U[r][col] / U[col][col];

    __syncthreads();  // make sure all threads have computed L[r][col]

    // distribute this loop across the y-dimension of the threadblock
    for (int k = col + threadIdx.y; k < dimension; k += blockDim.y) {
      U[r][k] = U[r][k] - L[r][col] * U[col][k];
    }
  }
}

// function to print a matrix
void print_matrix(double** matrix, int dimension) {
  for (int r = 0; r < dimension; r++) {
    for (int c = 0; c < dimension; c++) {
      printf("%f ", matrix[r][c]);
    }
    printf("\n");
  }
}

void LU_fact(double** matrix, double** L, double** U, double** P,
             int dimension) {
  // make sure that P,L = I and U = matrix
  for (int r = 0; r < dimension; r++) {
    for (int c = 0; c < dimension; c++) {
      if (r == c) {
        L[r][c] = 1;
        P[r][c] = 1;
      } else {
        L[r][c] = 0;
        P[r][c] = 0;
      }
      U[r][c] = matrix[r][c];
    }
  }

  // begin factorization with partial pivoting
  for (int c = 0; c < dimension - 1; c++) {
    double max = fabs(U[c][c]);
    int max_index = c;
    // find the max for the partial pivot
    for (int r = c; r < dimension - 1; r++) {
      if (fabs(U[r][c]) > max) {
        max = fabs(U[r][c]);
        max_index = r;
      }
    }

    swap_rows_U<<<1, 32>>>(c, max_index, c, dimension, U);

    swap_rows_L<<<1, 32>>>(c, max_index, c, L);

    swap_rows_P<<<1, 32>>>(c, max_index, dimension, P);

    row_ops_kernel<<<1, 32>>>(c, dimension, L, U);
    hipDeviceSynchronize();
  }
}

// cuda kernel to generate random values for a 2d matrix
// __global__ void generateRandomValues(double** matrix, int n, int seed,
//                                      curandState_t* state) {
//   int row = blockIdx.x * blockDim.x + threadIdx.x;
//   int col = blockIdx.y * blockDim.y + threadIdx.y;

//   // Fill diagonal with random values between 1 and 10
//   random_device rd;
//   mt19937 gen(rd());
//   uniform_real_distribution<double> dis(1.0, 10.0);

//   // Fill off-diagonal with random values between -1 and 1
//   uniform_real_distribution<double> dis_off(-1.0, 1.0);
//   if (row < n && col < n) {
//     // use rand to generate random values
//     if (row == col) {
//       matrix[row][col] = dis(gen);
//     } else {
//       matrix[row][col] = dis_off(gen);
//     }
//   }
// }

// cuda kernel to generate a strictly diagonally dominant matrix
__global__ void generateSDDMatrix(double** matrix, int n) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < n && col < n) {
    if (row == col) {
      int sum = 0;
      for (int i = 0; i < n; i++) {
        if (i != row) {
          sum += matrix[row][i];
        }
      }
      matrix[row][col] = sum + 1;
    }
  }
}

__global__ void generateRandomValues(double** matrix, int n,
                                     hiprandState* state) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  int index = row * n + col;

  if (col < n && row < n) {
    hiprandState localState = state[index];
    matrix[row][col] = hiprand_uniform(&localState) * 100.0;
    state[index] = localState;
  }
}

int main(int argc, char* argv[]) {
  // initialize matrix A using cudaMallocManaged
  int dimension = 3;
  double** A;
  double** L;
  double** U;
  double** P;
  double** LU;
  double** PA;
  hiprandState* state;
  bool* equal;

  hipMallocManaged(&A, dimension * sizeof(double*));
  hipMallocManaged(&L, dimension * sizeof(double*));
  hipMallocManaged(&U, dimension * sizeof(double*));
  hipMallocManaged(&P, dimension * sizeof(double*));
  hipMallocManaged(&LU, dimension * sizeof(double*));
  hipMallocManaged(&PA, dimension * sizeof(double*));
  hipMallocManaged(&equal, sizeof(bool));
  hipMallocManaged(&state, dimension * dimension * sizeof(hiprandState));

  for (int r = 0; r < dimension; r++) {
    hipMallocManaged(&A[r], dimension * sizeof(double));
    hipMallocManaged(&L[r], dimension * sizeof(double));
    hipMallocManaged(&U[r], dimension * sizeof(double));
    hipMallocManaged(&P[r], dimension * sizeof(double));
    hipMallocManaged(&LU[r], dimension * sizeof(double));
    hipMallocManaged(&PA[r], dimension * sizeof(double));
  }

  // initialize A to be this matrix
  // A = [ 2  1  1 ]
  //     [ 4  3  3 ]
  //     [ 8  7  9 ]
  // A[0][0] = 2;
  // A[0][1] = 1;
  // A[0][2] = 1;
  // A[1][0] = 4;
  // A[1][1] = 3;
  // A[1][2] = 3;
  // A[2][0] = 8;
  // A[2][1] = 7;
  // A[2][2] = 9;

  // initialize A to be a random matrix
  generateRandomValues<<<1, 32>>>(A, dimension, state);

  // generate a strictly diagonally dominant matrix
  // generateSDDMatrix<<<1, 32>>>(A, dimension);

  // print A
  printf("A = \n");
  print_matrix(A, dimension);

  // LU factorization
  // LU_fact(A, L, U, P, dimension);

  // // print results
  // printf("L = \n");
  // print_matrix(L, dimension);

  // printf("\nU = \n");
  // print_matrix(U, dimension);

  // // compute LU and PA
  // matrix_mult<<<1, 32>>>(L, U, LU, dimension);
  // // cudaDeviceSynchronize();
  // matrix_mult<<<1, 32>>>(P, A, PA, dimension);
  // cudaDeviceSynchronize();

  // // check if LU = PA using check_matrix_equivalence
  // check_matrix_equivalence<<<1, 32>>>(LU, PA, equal, dimension);
  // cudaDeviceSynchronize();

  // // print results
  // if (*equal) {
  //   printf("\nLU = PA\n");
  // } else {
  //   printf("\nLU != PA\n");
  // }

  // free memory
  for (int r = 0; r < dimension; r++) {
    hipFree(A[r]);
    hipFree(L[r]);
    hipFree(U[r]);
    hipFree(P[r]);
    hipFree(LU[r]);
    hipFree(PA[r]);
  }

  hipFree(A);
  hipFree(L);
  hipFree(U);
  hipFree(P);
  hipFree(LU);
  hipFree(PA);
  hipFree(equal);
  hipFree(state);

  return 0;
}